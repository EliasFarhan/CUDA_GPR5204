#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <dot.h>

#define BLOCK_SIZE 16

__global__ 
void dot(const float *a, const float *b, float *c, const int N)
{
	__shared__ float temp[BLOCK_SIZE];

	int index = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	if (index < N)
	{
		temp[threadIdx.x] = a[index] * b[index];
	}
	
	__syncthreads();

	if (threadIdx.x == 0)
	{
		float sum = 0.0f;
		for (int i = 0; i < BLOCK_SIZE; i++)
		{
			if (i < N)
			{
				sum += temp[i];
			}
		}
		atomicAdd(c, sum);
	}
}

float dotGpu(const float *hostV1, const float *hostV2, size_t len)
{
	float *devV1, *devV2, *devResult;

	// Allocate Unified Memory � accessible from CPU or GPU
	hipMallocManaged(&devV1, len * sizeof(float));
	hipMallocManaged(&devV2, len * sizeof(float));
	hipMallocManaged(&devResult, sizeof(float));
	*devResult = 0.0f;

	hipMemcpy(devV1, hostV1, len * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devV2, hostV2, len * sizeof(float), hipMemcpyHostToDevice);


	const int numBlocks = (len + BLOCK_SIZE - 1) / BLOCK_SIZE;

	dot <<<numBlocks, BLOCK_SIZE >>> (devV1, devV2, devResult, len);
	hipDeviceSynchronize();

	float v = *devResult;

	hipFree(devV1);
	hipFree(devV2);
	hipFree(devResult);
	return v;
}

void matrixMultGpu(const float* m1, const float* m2, float* result, const size_t newWidth, const size_t newHeight, const size_t len)
{
	float *devV1, *devV2, *devResult;
	hipMallocManaged(&devV1, len * newHeight * sizeof(float));
	hipMallocManaged(&devV2, len * newWidth * sizeof(float));
	hipMallocManaged(&devResult, newWidth*newHeight*sizeof(float));
	for (int i = 0; i < newWidth*newHeight; i++)
	{
		devResult[i] = 0.0f;
	}

	hipMemcpy(devV1, m1, len * newHeight * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devV2, m2, len * newWidth * sizeof(float), hipMemcpyHostToDevice);


	const int numBlocks = (len + BLOCK_SIZE - 1) / BLOCK_SIZE;
	
	for (int y = 0; y < newHeight; y++)
	{
		for (int x = 0; x < newWidth; x++)
		{
			const int xOrigin = x * len;
			const int yOrigin = y * len;
			dot <<<numBlocks, BLOCK_SIZE >>> (devV1+yOrigin, devV2+xOrigin, devResult+(x*newWidth + y), len);
		}
	}

	hipDeviceSynchronize();
	for (int i = 0; i < newWidth*newHeight; i++)
	{
		result[i] = devResult[i];
	}
	hipFree(devV1);
	hipFree(devV2);
	hipFree(devResult);
}