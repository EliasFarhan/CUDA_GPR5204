#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <dot.h>

#define BLOCK_SIZE 256

__global__ 
void dot(const float *a, const float *b, float *c, const int N)
{
	__shared__ float temp[BLOCK_SIZE];

	int index = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	if (index < N)
	{
		temp[threadIdx.x] = a[index] * b[index];
	}
	
	__syncthreads();

	if (threadIdx.x == 0)
	{
		float sum = 0.0f;
		for (int i = 0; i < BLOCK_SIZE; i++)
		{
			if (i < N)
			{
				sum += temp[i];
			}
		}
		atomicAdd(c, sum);
	}
}


float dotGpu(const float *hostV1, const float *hostV2, size_t len)
{
	float *devV1, *devV2, *devResult;

	// Allocate Unified Memory � accessible from CPU or GPU
	hipMallocManaged(&devV1, len * sizeof(float));
	hipMallocManaged(&devV2, len * sizeof(float));
	hipMallocManaged(&devResult, sizeof(float));
	*devResult = 0.0f;

	hipMemcpy(devV1, hostV1, len * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devV2, hostV2, len * sizeof(float), hipMemcpyHostToDevice);


	const int numBlocks = (len + BLOCK_SIZE - 1) / BLOCK_SIZE;

	dot <<<numBlocks, BLOCK_SIZE >>> (devV1, devV2, devResult, len);
	hipDeviceSynchronize();

	float v = *devResult;

	hipFree(devV1);
	hipFree(devV2);
	hipFree(devResult);
	return v;
}
