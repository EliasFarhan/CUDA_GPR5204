#include "hip/hip_runtime.h"
#include <addition.h>
#include <hip/hip_runtime.h>

__global__
void add(int n, const float *x, const float *y, float* result)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		result[i] = x[i] + y[i];
}

void GpuAdd(const float* x, const float* y, float* result, const size_t N)
{
	float* gpuX;
	float* gpuY;
	float* gpuResult;
	hipMallocManaged(&gpuX, N * sizeof(float));
	hipMallocManaged(&gpuY, N * sizeof(float));

	hipMallocManaged(&gpuResult, N * sizeof(float));


	hipMemcpy(gpuX, x, N, hipMemcpyHostToDevice);
	hipMemcpy(gpuY, y, N, hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	add << <numBlocks, blockSize >> > (N, gpuX, gpuY, gpuResult);
	hipDeviceSynchronize();
	hipMemcpy(gpuResult, result, N, hipMemcpyDeviceToHost);

	hipFree(gpuX);
	hipFree(gpuY);
	hipFree(gpuResult);
}